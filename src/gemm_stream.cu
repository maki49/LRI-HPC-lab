#include "hip/hip_runtime.h"
#include "gemm.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <string>
#include <fstream>
#include <iostream>
#include <chrono>
__global__ void print_matrix(double* A, int M, int N, int istream)
{
    printf("matrix C in stream %d: \n", istream);
    for (int r = 0; r < M; r++)
    {
        for (int c = 0;c < N;c++)
        {
            printf("%f ", A[c * M + r]);
        }
        printf("\n");
    }
}

template<>
void CudaGemm<double>::gemmblas_stream(std::vector<CaseDense<double>>ABC, int M, int N, int K, double alpha, double beta)
{
    // create cuda stream
    hipStream_t* streams = new hipStream_t[ABC.size()];
    for (int i = 0; i < ABC.size(); i++)
    {
        hipStreamCreate(&streams[i]);
    }
    // device memory
    double** d_As = new double* [ABC.size()];
    double** d_Bs = new double* [ABC.size()];
    double** d_Cs = new double* [ABC.size()];

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    auto start = std::chrono::system_clock::now();
    for (int i = 0;i < ABC.size();i++)
    {
        // auto start_i = std::chrono::system_clock::now();

        // why hipMallocAsync is slower than hipMalloc?
        // hipMallocAsync((void**)&d_As[i], M * K * sizeof(double), streams[i]);
        // hipMallocAsync((void**)&d_Bs[i], K * N * sizeof(double), streams[i]);
        // hipMallocAsync((void**)&d_Cs[i], M * N * sizeof(double), streams[i]);
        hipMalloc((void**)&d_As[i], M * K * sizeof(double));
        hipMalloc((void**)&d_Bs[i], K * N * sizeof(double));
        hipMalloc((void**)&d_Cs[i], M * N * sizeof(double));
        // hipMemcpy(d_As[i], ABC[i].A.data(), M * K * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpyAsync(d_Bs[i], ABC[i].B.data(), K * N * sizeof(double), hipMemcpyHostToDevice, streams[i]);
        
        hipblasSetStream(handle, streams[i]);
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_As[i], M, d_Bs[i], K, &beta, d_Cs[i], M);

        // print_matrix<<<1, 1, 0, streams[i]>>>(d_Cs[i], M, N, i);

        // hipMemcpy(ABC[i].C.data(), d_Cs[i], M * N * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpyAsync(ABC[i].C.data(), d_Cs[i], M * N * sizeof(double), hipMemcpyDeviceToHost, streams[i]);

        // auto end_i = std::chrono::system_clock::now();
        // std::chrono::duration<double> duration_i = end_i - start_i;
        // std::cout << "stream:" << i << "time elapsed: " << double(duration_i.count()) << " ms" << std::endl;

        // hipFreeAsync(d_As[i], streams[i]);
        // hipFreeAsync(d_Bs[i], streams[i]);
        // hipFreeAsync(d_Cs[i], streams[i]);
        hipFree(d_As[i]);
        hipFree(d_Bs[i]);
        hipFree(d_Cs[i]);
    }

    hipblasDestroy(handle);
    for (int i = 0; i < ABC.size(); i++)
    {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "total time elapsed(GPU): " << double(duration.count()) << " ms" << std::endl;

    delete[] streams;
    delete[] d_As;
    delete[] d_Bs;
    delete[] d_Cs;
}


template<>
void CudaGemm<double>::gemmblas_cpu_ref(std::vector<CaseDense<double>>ABC, int M, int N, int K, double alpha, double beta)
{
    auto start = std::chrono::system_clock::now();
    for (int i = 0;i < ABC.size();i++)
        dgemm_("N", "N", &M, &N, &K, &alpha, ABC[i].A.data(), &M, ABC[i].B.data(), &K, &beta, ABC[i].C.data(), &M);
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "total time elapsed(CPU): " << double(duration.count()) << " ms" << std::endl;

}