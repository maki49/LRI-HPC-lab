#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

static const char* _cublasGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
    case HIPBLAS_STATUS_SUCCESS:
        return "CUBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
        return "CUBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
        return "CUBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
        return "CUBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
        return "CUBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
        return "CUBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
        return "CUBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
        return "CUBLAS_STATUS_INTERNAL_ERROR";
    default:
        return "Unknown";
    }
}


static const char* _cusparseGetErrorEnum(hipsparseStatus_t error) {
    switch (error) {
    case HIPSPARSE_STATUS_SUCCESS:
        return "CUSPARSE_STATUS_SUCCESS";

    case HIPSPARSE_STATUS_NOT_INITIALIZED:
        return "CUSPARSE_STATUS_NOT_INITIALIZED";

    case HIPSPARSE_STATUS_ALLOC_FAILED:
        return "CUSPARSE_STATUS_ALLOC_FAILED";

    case HIPSPARSE_STATUS_INVALID_VALUE:
        return "CUSPARSE_STATUS_INVALID_VALUE";

    case HIPSPARSE_STATUS_ARCH_MISMATCH:
        return "CUSPARSE_STATUS_ARCH_MISMATCH";

    case HIPSPARSE_STATUS_MAPPING_ERROR:
        return "CUSPARSE_STATUS_MAPPING_ERROR";

    case HIPSPARSE_STATUS_EXECUTION_FAILED:
        return "CUSPARSE_STATUS_EXECUTION_FAILED";

    case HIPSPARSE_STATUS_INTERNAL_ERROR:
        return "CUSPARSE_STATUS_INTERNAL_ERROR";

    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "CUSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }

    return "<unknown>";
}

void cublasAssert(hipblasStatus_t code, const char* file, int line, bool abort = true) {
    if (code != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Assert: %s %s %d\n", _cublasGetErrorEnum(code), file, line);
        if (abort) exit(code);
    }
}

void cusparseAssert(hipsparseStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPSPARSE_STATUS_SUCCESS)
    {
        fprintf(stderr, "CUSPARSE Assert: %s %s %d\n", _cusparseGetErrorEnum(code), file, line);
        if (abort) exit(code);
    }
}

template<typename T>
void alloc_h2d(const T* h, T*& d, const int size)
{
    hipMalloc((void**)&d, size * sizeof(T));
    hipMemcpy(d, h, size * sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
void alloc_d2h(const T* d, T*& h, const int size)
{
    h = new T[size];
    hipMemcpy(h, d, size * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void d2h(const T* d, T* h, const int size)
{
    hipMemcpy(h, d, size * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void ad2h(const T* d, T* h, const int size, hipStream_t stream)
{
    hipMemcpyAsync(h, d, size * sizeof(T), hipMemcpyDeviceToHost, stream);
}

template<typename T>
void h2d(const T* h, T* d, const int size)
{
    hipMemcpy(d, h, size * sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
void ah2d(const T* h, T* d, const int size, hipStream_t stream)
{
    hipMemcpyAsync(d, h, size * sizeof(T), hipMemcpyHostToDevice, stream);
}

template<typename T>
void dfree(T* d)
{
    hipFree(d);
}


//int
template void alloc_h2d<int>(const int* h, int*& d, const int size);
template void alloc_d2h<int>(const int* d, int*& h, const int size);
template void d2h<int>(const int* d, int* h, const int size);
template void ad2h<int>(const int* d, int* h, const int size, hipStream_t stream);
template void h2d<int>(const int* h, int* d, const int size);
template void ah2d<int>(const int* h, int* d, const int size, hipStream_t stream);
template void dfree<int>(int* d);

template void alloc_h2d<unsigned long>(const unsigned long* h, unsigned long*& d, const int size);
template void alloc_d2h<unsigned long>(const unsigned long* d, unsigned long*& h, const int size);
template void d2h<unsigned long>(const unsigned long* d, unsigned long* h, const int size);
template void ad2h<unsigned long>(const unsigned long* d, unsigned long* h, const int size, hipStream_t stream);
template void h2d<unsigned long>(const unsigned long* h, unsigned long* d, const int size);
template void ah2d<unsigned long>(const unsigned long* h, unsigned long* d, const int size, hipStream_t stream);
template void dfree<unsigned long>(unsigned long* d);


//double
template void alloc_h2d<double>(const double* h, double*& d, const int size);
template void alloc_d2h<double>(const double* d, double*& h, const int size);
template void d2h<double>(const double* d, double* h, const int size);
template void ad2h<double>(const double* d, double* h, const int size, hipStream_t stream);
template void h2d<double>(const double* h, double* d, const int size);
template void ah2d<double>(const double* h, double* d, const int size, hipStream_t stream);
template void dfree<double>(double* d);