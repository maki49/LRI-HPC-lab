#include "hip/hip_runtime.h"
#include "gemm.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <iostream>
#include <assert.h>

#define cusparseErrcheck(err) {cusparseAssert(err, __FILE__, __LINE__);}

static const char* _cusparseGetErrorEnum(hipsparseStatus_t error) {
    switch (error) {
    case HIPSPARSE_STATUS_SUCCESS:
        return "HIPSPARSE_STATUS_SUCCESS";

    case HIPSPARSE_STATUS_NOT_INITIALIZED:
        return "HIPSPARSE_STATUS_NOT_INITIALIZED";

    case HIPSPARSE_STATUS_ALLOC_FAILED:
        return "HIPSPARSE_STATUS_ALLOC_FAILED";

    case HIPSPARSE_STATUS_INVALID_VALUE:
        return "HIPSPARSE_STATUS_INVALID_VALUE";

    case HIPSPARSE_STATUS_ARCH_MISMATCH:
        return "HIPSPARSE_STATUS_ARCH_MISMATCH";

    case HIPSPARSE_STATUS_MAPPING_ERROR:
        return "HIPSPARSE_STATUS_MAPPING_ERROR";

    case HIPSPARSE_STATUS_EXECUTION_FAILED:
        return "HIPSPARSE_STATUS_EXECUTION_FAILED";

    case HIPSPARSE_STATUS_INTERNAL_ERROR:
        return "HIPSPARSE_STATUS_INTERNAL_ERROR";

    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }

    return "<unknown>";
}

inline void cusparseAssert(hipsparseStatus_t code, const char* file, int line, bool abort = true)
{
    if (code != HIPSPARSE_STATUS_SUCCESS)
    {
        fprintf(stderr, "CUSPARSE Assert: %s %s %d\n", _cusparseGetErrorEnum(code), file, line);
        if (abort) exit(code);
    }
}

inline void alloc_h2d(const int* h, int*& d, const int size)
{
    hipMalloc((void**)&d, size * sizeof(int));
    hipMemcpy(d, h, size * sizeof(int), hipMemcpyHostToDevice);
}
inline void alloc_h2d(const double* h, double*& d, const int size)
{
    hipMalloc((void**)&d, size * sizeof(double));
    hipMemcpy(d, h, size * sizeof(double), hipMemcpyHostToDevice);
}

inline void alloc_d2h(const int* d, int*& h, const int size)
{
    h = new int[size];
    hipMemcpy(h, d, size * sizeof(int), hipMemcpyDeviceToHost);
}
inline void alloc_d2h(const double* d, double*& h, const int size)
{
    h = new double[size];
    hipMemcpy(h, d, size * sizeof(double), hipMemcpyDeviceToHost);
}

inline void d2h(const int* d, int* h, const int size)
{
    hipMemcpy(h, d, size * sizeof(int), hipMemcpyDeviceToHost);
}
inline void d2h(const double* d, double* h, const int size)
{
    hipMemcpy(h, d, size * sizeof(double), hipMemcpyDeviceToHost);
}

inline void h2d(const int* h, int* d, const int size)
{
    hipMemcpy(d, h, size * sizeof(int), hipMemcpyHostToDevice);
}
inline void h2d(const double* h, double* d, const int size)
{
    hipMemcpy(d, h, size * sizeof(double), hipMemcpyHostToDevice);
}

inline void dfree(int* d)
{
    hipFree(d);
}
inline void dfree(double* d)
{
    hipFree(d);
}

__device__ double get_element(const double* A, int r, int c, int lda)
{
    return A[c * lda + r];
}

__device__ void set_element(double* A, int r, int c, int lda, const double val)
{
    A[c * lda + r] = val;
}

__host__ double get_element_h(const double* A, int r, int c, int lda)
{
    return A[c * lda + r];
}

__host__ void set_element_h(double* A, int r, int c, int lda, const double val)
{
    A[c * lda + r] = val;
}



__global__ void gemm2d_kernel(double* A, double* B, double* C, int M, int N, int K, double alpha, double beta)
{
    // 2d-structure C(M, N)
    // each thread computes one element of C
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N)
    {
        double sum = 0;
        for (int i = 0; i < K; i++)
        {
            sum += get_element(A, row, i, M) * get_element(B, i, col, K);
        }
        set_element(C, row, col, M, alpha * sum + beta * get_element(C, row, col, M));
    }
}

template<typename T>
void set_zeros(T* A, const int size)
{
    hipMemset(A, 0, size * sizeof(T));
}

template<>
void CudaGemm<double>::gemm2d(double* A, double* B, double* C, int M, int N, int K, double alpha, double beta)
{
    int bx = 32;
    int by = 32;
    int gx = (M - 1) / bx + 1;
    int gy = (N - 1) / by + 1;
    dim3 block(bx, by);
    dim3 grid(gx, gy);

    // allocate device memory
    double* dA;
    double* dB;
    double* dC;
    alloc_h2d(A, dA, M * K);
    alloc_h2d(B, dB, K * N);
    alloc_h2d(C, dC, M * N);
    gemm2d_kernel << <grid, block >> > (dA, dB, dC, M, N, K, alpha, beta);
    d2h(dC, C, M * N);
    dfree(dA);
    dfree(dB);
    dfree(dC);
}

template<>
void CudaGemm<double>::gemmblas(double* A, double* B, double* C, int M, int N, int K, double alpha, double beta)
{
    // allocate device memory
    double* dA;
    double* dB;
    double* dC;
    alloc_h2d(A, dA, M * K);
    alloc_h2d(B, dB, K * N);
    alloc_h2d(C, dC, M * N);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, dA, M, dB, K, &beta, dC, M);
    hipblasDestroy(handle);

    d2h(dC, C, M * N);
    dfree(dA);
    dfree(dB);
    dfree(dC);
}

template<typename T>
int CudaGemm<T>::dense2csr(const T* A, const int M, const int N, T* V, int* CI, int* RI, const T thr)
{
    int nnz = 0;
    for (int i = 0; i < M; i++)
    {
        int row_nnz = 0;
        for (int j = 0; j < N; j++)
        {
            T v = get_element_h(A, i, j, M);
            if (v > thr)
            {
                V[nnz] = v;
                CI[nnz] = j;
                nnz++;
                row_nnz++;
            }
        }
        RI[i + 1] = nnz;
        assert(RI[i + 1] == RI[i] + row_nnz);
    }
    return nnz;
}

template<typename T>
int CudaGemm<T>::dense2csr(const T* A, const int M, const int N, std::vector<T>& V, std::vector<int>& CI, std::vector<int>& RI, const T thr)
{
    RI.resize(M + 1);
    int nnz = 0;
    for (int i = 0; i < M; i++)
    {
        int row_nnz = 0;
        for (int j = 0; j < N; j++)
        {
            T v = get_element_h(A, i, j, M);
            if (v > thr)
            {
                V.push_back(v);
                CI.push_back(j);
                nnz++;
                row_nnz++;
            }
        }
        RI[i + 1] = nnz;
        assert(RI[i + 1] == RI[i] + row_nnz);
    }
    assert(nnz==V.size() && nnz==CI.size());
    return nnz;
}

template<typename T>
void CudaGemm<T>::csr2dense(const T* V, const int* CI, const int* RI, const int M, const int N, T* A)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = RI[i]; j < RI[i + 1]; j++)
        {
            set_element_h(A, i, CI[j], M, V[j]);
        }
    }
}
template<>
void CudaGemm<double>::gemmsparse_csr(double* A, double* B, double* C, int M, int N, int K, double alpha, double beta)
{

    // reference: https://docs.nvidia.com/cuda/cusparse/#cusparsespgemm
    // =============host==============
    
    std::vector<int> hRowPtrA(M + 1);
    std::vector<int> hColIndA;
    std::vector<double> hValA;
    std::vector<int> hRowPtrB(K + 1);
    std::vector<int> hColIndB;
    std::vector<double> hValB;
    std::vector<int> hRowPtrC(M + 1);
    std::vector<int> hColIndC;
    std::vector<double> hValC;


    // convert dense to CSR
    int nnzA = dense2csr(A, M, K, hValA, hColIndA, hRowPtrA);
    int nnzB = dense2csr(B, K, N, hValB, hColIndB, hRowPtrB);

    // =============device==============
    int* dRowPtrA;
    int* dColIndA;
    double* dValA;
    int* dRowPtrB;
    int* dColIndB;
    double* dValB;
    int* dRowPtrC;
    int* dColIndC;
    double* dValC;

    
    alloc_h2d(hRowPtrA.data(), dRowPtrA, M + 1);
    alloc_h2d(hColIndA.data(), dColIndA, nnzA);
    alloc_h2d(hValA.data(), dValA, nnzA);
    alloc_h2d(hRowPtrB.data(), dRowPtrB, K + 1);
    alloc_h2d(hColIndB.data(), dColIndB, nnzB);
    alloc_h2d(hValB.data(), dValB, nnzB);
    alloc_h2d(hRowPtrC.data(), dRowPtrC, M + 1);
    // alloc_h2d(hColIndC.data(), dColIndC, 0);
    // alloc_h2d(hValC.data(), dValC, 0);

    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseIndexType_t i32 = HIPSPARSE_INDEX_32I;
    hipsparseIndexBase_t b0 = HIPSPARSE_INDEX_BASE_ZERO;
    hipDataType r64f = HIP_R_64F;
    hipsparseSpGEMMAlg_t alg = HIPSPARSE_SPGEMM_DEFAULT;


    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    hipsparseSpMatDescr_t matA, matB, matC;
    hipsparseCreateCsr(&matA, M, K, nnzA, dRowPtrA, dColIndA, dValA,
        i32, i32, b0, r64f);
    hipsparseCreateCsr(&matB, K, N, nnzB, dRowPtrB, dColIndB, dValB,
        i32, i32, b0, r64f);
    hipsparseCreateCsr(&matC, M, N, 0, dRowPtrC, NULL, NULL,
        i32, i32, b0, r64f);

    // cannot pass a NULL-dRowPtrC to hipsparseCreateCsr
    // hipsparseCreateCsr(&matC, M, N, 0, NULL, NULL, NULL,
    //     i32, i32, b0, r64f);

    hipsparseSpGEMMDescr_t spgemmDesc;
    cusparseErrcheck(hipsparseSpGEMM_createDescr(&spgemmDesc));
    // ask buffersize bytes for external memory
    void* dbuffer1 = NULL, * dbuffer2 = NULL;
    size_t buffersize1 = 0, buffersize2 = 0;
    cusparseErrcheck(hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA, matB, &beta, matC, r64f, alg, spgemmDesc, &buffersize1, NULL));
    hipMalloc(&dbuffer1, buffersize1);
    // inspect the matrices A and B to understand the memory requiremnets for the next step
    cusparseErrcheck(hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA, matB, &beta, matC, r64f, alg, spgemmDesc, &buffersize1, dbuffer1));
    // ask buffersize2 bytes for external memory
    cusparseErrcheck(hipsparseSpGEMM_compute(handle, opA, opB, &alpha, matA, matB, &beta, matC, r64f, alg, spgemmDesc, &buffersize2, NULL));
    hipMalloc(&dbuffer2, buffersize2);
    //compute A*B
    cusparseErrcheck(hipsparseSpGEMM_compute(handle, opA, opB, &alpha, matA, matB, &beta, matC, r64f, alg, spgemmDesc, &buffersize2, dbuffer2));
    // get C's non-zero elements: nnzC1
    
    // copy C back to host
    int64_t nrowC, ncolC, nnzC;
    cusparseErrcheck(hipsparseSpMatGetSize(matC, &nrowC, &ncolC, &nnzC));

    // allocate device memory for C
    hipMalloc((void**)&dColIndC, nnzC * sizeof(int));
    hipMalloc((void**)&dValC, nnzC * sizeof(double));

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy

    // update C with the new pointer
    cusparseErrcheck(hipsparseCsrSetPointers(matC, dRowPtrC, dColIndC, dValC));

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to C
    cusparseErrcheck(hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matB, &beta, matC, r64f, alg, spgemmDesc));

    // destroy matrix/vector descriptors
    cusparseErrcheck(hipsparseSpGEMM_destroyDescr(spgemmDesc));
    cusparseErrcheck(hipsparseDestroySpMat(matA));
    cusparseErrcheck(hipsparseDestroySpMat(matB));
    cusparseErrcheck(hipsparseDestroySpMat(matC));
    cusparseErrcheck(hipsparseDestroy(handle));
    
    // copy C (CSR) from device to host
    hColIndC.resize(nnzC);
    hValC.resize(nnzC);
    d2h(dRowPtrC, hRowPtrC.data(), M + 1);
    d2h(dColIndC, hColIndC.data(), nnzC);
    d2h(dValC, hValC.data(), nnzC);

    // free device memory
    hipFree(dRowPtrA);
    hipFree(dColIndA);
    hipFree(dValA);
    hipFree(dRowPtrB);
    hipFree(dColIndB);
    hipFree(dValB);
    hipFree(dRowPtrC);
    hipFree(dColIndC);
    hipFree(dValC);
    

    // convert C (CSR) to dense
    csr2dense(hValC.data(), hColIndC.data(), hRowPtrC.data(), M, N, C);
    
}
template class CudaGemm<double>;